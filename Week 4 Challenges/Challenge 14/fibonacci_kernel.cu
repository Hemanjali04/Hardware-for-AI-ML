#include <hip/hip_runtime.h>

__device__ int fibonacci(int n) {
    if (n == 0) return 0;
    if (n == 1) return 1;
    int a = 0, b = 1, c;
    for (int i = 2; i <= n; i++) {
        c = a + b;
        a = b;
        b = c;
    }
    return b;
}

__global__ void fibonacci_cuda(int* d_result, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        d_result[idx] = fibonacci(idx);
    }
}
