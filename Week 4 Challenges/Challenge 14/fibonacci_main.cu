#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

void fibonacci_cpu(int* result, int N);
__global__ void fibonacci_cuda(int* d_result, int N);

int main() {
    const int N = 220;

    // CPU
    int* cpu_result = new int[N];
    auto start_cpu = std::chrono::high_resolution_clock::now();
    fibonacci_cpu(cpu_result, N);
    auto end_cpu = std::chrono::high_resolution_clock::now();

    // GPU
    int* h_result = new int[N];
    int* d_result;
    hipMalloc(&d_result, N * sizeof(int));

    auto start_gpu = std::chrono::high_resolution_clock::now();
    fibonacci_cuda<<<(N + 255) / 256, 256>>>(d_result, N);
    hipDeviceSynchronize();
    hipMemcpy(h_result, d_result, N * sizeof(int), hipMemcpyDeviceToHost);
    auto end_gpu = std::chrono::high_resolution_clock::now();

    // Verify
    bool match = true;
    for (int i = 0; i < N; i++) {
        if (cpu_result[i] != h_result[i]) {
            match = false;
            std::cout << "Mismatch at " << i << ": CPU = " << cpu_result[i] << ", GPU = " << h_result[i] << "\n";
            break;
        }
    }

    std::chrono::duration<double> cpu_time = end_cpu - start_cpu;
    std::chrono::duration<double> gpu_time = end_gpu - start_gpu;

    std::cout << "CPU Time: " << cpu_time.count() << "s\n";
    std::cout << "GPU Time: " << gpu_time.count() << "s\n";
    std::cout << "Results match: " << (match ? "YES" : "NO") << "\n";

    hipFree(d_result);
    delete[] cpu_result;
    delete[] h_result;

    return 0;
}
